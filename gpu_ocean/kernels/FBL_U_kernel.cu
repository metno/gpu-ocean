#include "hip/hip_runtime.h"
/*
This OpenCL kernel implements part of the Forward Backward Linear 
numerical scheme for the shallow water equations, described in 
L. P. Røed, "Documentation of simple ocean models for use in ensemble
predictions", Met no report 2012/3 and 2012/5 .

Copyright (C) 2016  SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "common.cu"

// Finds the coriolis term based on the linear Coriolis force
// f = \tilde{f} + beta*(y-y0)
__device__ float linear_coriolis_term(const float f, const float beta,
			   const float tj, const float dy,
			   const float y_zero_reference_cell) {
    // y_0 is at the southern face of the row y_zero_reference_cell.
    float y = (tj-y_zero_reference_cell + 0.5f)*dy;
    return f + beta * y;
}



/**
  * Kernel that evolves U one step in time.
  */
extern "C" {
__global__ void computeUKernel(
        //Discretization parameters
        int nx_, int ny_,
        float dx_, float dy_, float dt_,
    
        //Physical parameters
        float g_, //< Gravitational constant
        float f_, //< Coriolis coefficient
        float beta_, //< Coriolis force f_ + beta_*(y-y0)
        float y_zero_reference_cell_, // the cell row representing y0 (y0 at southern face)
        float r_, //< Bottom friction coefficient
    
        //Data
        float* H_ptr_, int H_pitch_,
        float* U_ptr_, int U_pitch_,
        float* V_ptr_, int V_pitch_,
        float* eta_ptr_, int eta_pitch_,
    
        // Wind stress parameters
        float wind_stress_t_) {
    
    __shared__ float H_shared[block_height][block_width+1];
    __shared__ float V_shared[block_height+1][block_width+1];
    __shared__ float eta_shared[block_height][block_width+1];

    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    //Index of block within domain
    const int bx = blockDim.x * blockIdx.x;
    const int by = blockDim.y * blockIdx.y;

    //Index of cell within domain
    const int ti = bx + tx;
    const int tj = by + ty;
    
    //Compute pointer to row "tj" in the U array
    float* const U_row = (float*) ((char*) U_ptr_ + U_pitch_*tj);

    //Read current U
    float U_current = 0.0f;
    if (ti < nx_ + 1 && tj < ny_) {
        U_current = U_row[ti];
    }

    //Read H and eta into local memory
    for (int j=ty; j<block_height; j+=blockDim.y) {
        const int l = by + j;
        
        //Compute the pointer to row "l" in the H and eta arrays
        float* const H_row = (float*) ((char*) H_ptr_ + H_pitch_*l);
        float* const eta_row = (float*) ((char*) eta_ptr_ + eta_pitch_*l);
        
        for (int i=tx; i<block_width+1; i+=blockDim.x) {
            const int k = bx + i - 1;
            
            if (k >= 0 && k < nx_ && l < ny_) {
                H_shared[j][i] = H_row[k];
                eta_shared[j][i] = eta_row[k];
            }
            else {
                H_shared[j][i] = 0.0f;
                eta_shared[j][i] = 0.0f;
            }
        }
    }

    //Read V into shared memory
    for (int j=ty; j<block_height+1; j+=blockDim.y) {
        const int l = by + j;
        
        //Compute the pointer to current row in the V array
        float* const V_row = (float*) ((char*) V_ptr_ + V_pitch_*l);
        
        for (int i=tx; i<block_width+1; i+=blockDim.x) {
            const int k = bx + i - 1;
            
            if (k >= 0 && k < nx_ && l < ny_+1) {
                V_shared[j][i] = V_row[k];
            }
            else {
                V_shared[j][i] = 0.0f;
            }
        }
    }

    //Make sure all threads have read into shared mem
    __syncthreads();

    //Reconstruct H at the U position
    float H_m = 0.5f*(H_shared[ty][tx] + H_shared[ty][tx+1]);

    // Coriolis forces at U position and V positions
    float f_u =   linear_coriolis_term(f_, beta_, tj,      dy_, y_zero_reference_cell_);
    float f_v_p = linear_coriolis_term(f_, beta_, tj+0.5f, dy_, y_zero_reference_cell_);
    float f_v_m = linear_coriolis_term(f_, beta_, tj-0.5f, dy_, y_zero_reference_cell_);
    
    //Reconstruct f*V at the U position
    float fV_m = 0.0f;
    if (tj==0) {
	// Using Coriolis at U position
        fV_m = 0.5f*f_u*(V_shared[ty+1][tx] + V_shared[ty+1][tx+1]);
    }
    else if (tj==ny_-1) {
	// Using Coriolis at U position
        fV_m = 0.5f*f_u*(V_shared[ty][tx] + V_shared[ty][tx+1]);
    }
    else {
        fV_m = 0.25f*( f_v_m*(V_shared[ty  ][tx] + V_shared[ty  ][tx+1])
		     + f_v_p*(V_shared[ty+1][tx] + V_shared[ty+1][tx+1]) );
    }

    //Calculate the friction coefficient
    float B = H_m/(H_m + r_*dt_);

    //Calculate the gravitational effect
    float P = g_*H_m*(eta_shared[ty][tx] - eta_shared[ty][tx+1])/dx_;
    
    //FIXME Check coordinates (ti_, tj_) here!!!
    //TODO Check coordinates (ti_, tj_) here!!!
    //WARNING Check coordinates (ti_, tj_) here!!!
    float X = windStressX(wind_stress_t_, ti, tj+0.5, nx_, ny_);

    //Compute the U at the next timestep
    float U_next = B*(U_current + dt_*(fV_m + P + X) );

    //Write to main memory for internal cells
    if (ti > 0 && ti < nx_ && tj < ny_) {
        U_row[ti] = U_next;
    }

    // TODO:
    // Currently, boundary conditions are individual kernels.
    // They should be moved to be within-kernel functions.

}
} // extern "C"