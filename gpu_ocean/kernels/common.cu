#include "hip/hip_runtime.h"
#ifndef COMMON_CU
#define COMMON_CU

#define _180_OVER_PI 57.29578f
#define PI_OVER_180 0.01745329f

/*
This file implements different helper functions etc.

Copyright (C) 2016, 2017, 2018 SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


inline __device__ float3 operator*(const float &a, const float3 &b) {
    return make_float3(a*b.x, a*b.y, a*b.z);
}

inline __device__ float3 operator/(const float3 &a, const float &b) {
    return make_float3(a.x/b, a.y/b, a.z/b);
}

inline __device__ float3 operator-(const float3 &a, const float3 &b) {
    return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

inline __device__ float3 operator+(const float3 &a, const float3 &b) {
    return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

inline __device__ __host__ float clamp(float f, float a, float b) {
    return fmaxf(a, fminf(f, b));
}

/**
  * Reads a block of data  with one ghost cell for the shallow water equations
  */
__device__ void readBlock1(float* h_ptr_, int h_pitch_,
                float* hu_ptr_, int hu_pitch_,
                float* hv_ptr_, int hv_pitch_,
                float Q[3][block_height+2][block_width+2], 
                const int nx_, const int ny_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    //Index of block within domain
    const int bx = blockIdx.x * blockDim.x;
    const int by = blockIdx.y * blockDim.y;
    
    //Read into shared memory
    for (int j=ty; j<block_height+2; j+=blockDim.y) {
        const int l = clamp(by + j, 0, ny_+1); // Out of bounds
        
        //Compute the pointer to current row in the arrays
        float* const h_row = (float*) ((char*) h_ptr_ + h_pitch_*l);
        float* const hu_row = (float*) ((char*) hu_ptr_ + hu_pitch_*l);
        float* const hv_row = (float*) ((char*) hv_ptr_ + hv_pitch_*l);
        
        for (int i=tx; i<block_width+2; i+=blockDim.x) {
            const int k = clamp(bx + i, 0, nx_+1); // Out of bounds
            
            Q[0][j][i] = h_row[k];
            Q[1][j][i] = hu_row[k];
            Q[2][j][i] = hv_row[k];
        }
    }
}





/**
  * Reads a block of data  with two ghost cells for the shallow water equations
  */
__device__ void readBlock2(float* h_ptr_, int h_pitch_,
                float* hu_ptr_, int hu_pitch_,
                float* hv_ptr_, int hv_pitch_,
                float Q[3][block_height+4][block_width+4], 
                const int nx_, const int ny_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    //Index of block within domain
    const int bx = blockIdx.x * blockDim.x;
    const int by = blockIdx.y * blockDim.y;
    
    //Read into shared memory
    for (int j=ty; j<block_height+4; j+=blockDim.y) {
        const int l = clamp(by + j, 0, ny_+3); // Out of bounds
        
        //Compute the pointer to current row in the arrays
        float* const h_row = (float*) ((char*) h_ptr_ + h_pitch_*l);
        float* const hu_row = (float*) ((char*) hu_ptr_ + hu_pitch_*l);
        float* const hv_row = (float*) ((char*) hv_ptr_ + hv_pitch_*l);
        
        for (int i=tx; i<block_width+4; i+=blockDim.x) {
            const int k = clamp(bx + i, 0, nx_+3); // Out of bounds
            
            Q[0][j][i] = h_row[k];
            Q[1][j][i] = hu_row[k];
            Q[2][j][i] = hv_row[k];
        }
    }
}

/**
  * Reads a block of data  with two ghost cells for the shallow water equations
  */
__device__ void readBlock2single(float* data_ptr_, int data_pitch_,
		      float shmem[block_height+4][block_width+4],
		      const int nx_, const int ny_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    //Index of block within domain
    const int bx = blockIdx.x * blockDim.x;
    const int by = blockIdx.y * blockDim.y;
    
    //Read into shared memory
    for (int j=ty; j<block_height+4; j+=blockDim.y) {
        const int l = clamp(by + j, 0, ny_+3); // Out of bounds
        
        //Compute the pointer to current row in the arrays
        float* const data_row = (float*) ((char*) data_ptr_ + data_pitch_*l);
        
        for (int i=tx; i<block_width+4; i+=blockDim.x) {
            const int k = clamp(bx + i, 0, nx_+3); // Out of bounds
	    shmem[j][i] = data_row[k];
        }
    }
}



/**
  * Writes a block of data to global memory for the shallow water equations.
  */
__device__ void writeBlock1(float* h_ptr_, int h_pitch_,
                 float* hu_ptr_, int hu_pitch_,
                 float* hv_ptr_, int hv_pitch_,
                 float Q[3][block_height+2][block_width+2],
                 const int nx_, const int ny_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    //Index of cell within domain
    const int ti = blockIdx.x * blockDim.x + threadIdx.x + 1; //Skip global ghost cells, i.e., +1
    const int tj = blockIdx.y * blockDim.y + threadIdx.y + 1;
    
    //Only write internal cells
    if (ti > 0 && ti < nx_+1 && tj > 0 && tj < ny_+1) {
        const int i = tx + 1; //Skip local ghost cells, i.e., +1
        const int j = ty + 1;

        float* const h_row  = (float*) ((char*) h_ptr_ + h_pitch_*tj);
        float* const hu_row = (float*) ((char*) hu_ptr_ + hu_pitch_*tj);
        float* const hv_row = (float*) ((char*) hv_ptr_ + hv_pitch_*tj);
        
        h_row[ti]  = Q[0][j][i];
        hu_row[ti] = Q[1][j][i];
        hv_row[ti] = Q[2][j][i];
    }
}





/**
  * Writes a block of data to global memory for the shallow water equations.
  */
__device__ void writeBlock2(float* h_ptr_, int h_pitch_,
                 float* hu_ptr_, int hu_pitch_,
                 float* hv_ptr_, int hv_pitch_,
                 float Q[3][block_height+4][block_width+4], 
                 const int nx_, const int ny_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    //Index of cell within domain
    const int ti = blockIdx.x * blockDim.x + threadIdx.x + 2; //Skip global ghost cells, i.e., +2
    const int tj = blockIdx.y * blockDim.y + threadIdx.y + 2;
    
    //Only write internal cells
    if (ti > 1 && ti < nx_+2 && tj > 1 && tj < ny_+2) {
        const int i = tx + 2; //Skip local ghost cells, i.e., +2
        const int j = ty + 2;

        float* const h_row  = (float*) ((char*) h_ptr_ + h_pitch_*tj);
        float* const hu_row = (float*) ((char*) hu_ptr_ + hu_pitch_*tj);
        float* const hv_row = (float*) ((char*) hv_ptr_ + hv_pitch_*tj);
        
        h_row[ti]  = Q[0][j][i];
        hu_row[ti] = Q[1][j][i];
        hv_row[ti] = Q[2][j][i];
    }
}






/**
  * No flow boundary conditions for the shallow water equations
  * with one ghost cell in each direction
  */
__device__ void noFlowBoundary1(float Q[3][block_height+2][block_width+2], const int nx_, const int ny_) {
    //Global index
    const int ti = blockIdx.x * blockDim.x + threadIdx.x + 1; //Skip global ghost cells, i.e., +1
    const int tj = blockIdx.y * blockDim.y + threadIdx.y + 1;
    
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    const int i = tx + 1; //Skip local ghost cells, i.e., +1
    const int j = ty + 1;
    
    //Fix boundary conditions
    if (ti == 1) {
        Q[0][j][i-1] =  Q[0][j][i];
        Q[1][j][i-1] = -Q[1][j][i];
        Q[2][j][i-1] =  Q[2][j][i];
    }
    if (ti == nx_) {
        Q[0][j][i+1] =  Q[0][j][i];
        Q[1][j][i+1] = -Q[1][j][i];
        Q[2][j][i+1] =  Q[2][j][i];
    }
    if (tj == 1) {
        Q[0][j-1][i] =  Q[0][j][i];
        Q[1][j-1][i] =  Q[1][j][i];
        Q[2][j-1][i] = -Q[2][j][i];
    }
    if (tj == ny_) {
        Q[0][j+1][i] =  Q[0][j][i];
        Q[1][j+1][i] =  Q[1][j][i];
        Q[2][j+1][i] = -Q[2][j][i];
    }
}




/**
  * No flow boundary conditions for the shallow water equations
  * with two ghost cells in each direction
  *
  * BC values are defined as follows: 
  * 1: Wall boundary condition
  * 2: Periodic boundary condition
  * 3: Open boundary (numerical sponge)
  */
__device__ void noFlowBoundary2Mix(float Q[3][block_height+4][block_width+4],
			const int nx_, const int ny_,
			const int bc_north_, const int bc_east_,
			const int bc_south_, const int bc_west_) {
    
    //Global index
    const int ti = blockIdx.x * blockDim.x + threadIdx.x + 2; //Skip global ghost cells, i.e., +2
    const int tj = blockIdx.y * blockDim.y + threadIdx.y + 2;
    
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    const int i = tx + 2; //Skip local ghost cells, i.e., +2
    const int j = ty + 2;
    
    if (ti == 2 && bc_west_ == 1) {
	// Wall boundary on west
	Q[0][j][i-1] =  Q[0][j][i];
	Q[1][j][i-1] = -Q[1][j][i];
	Q[2][j][i-1] =  Q[2][j][i];
        
	Q[0][j][i-2] =  Q[0][j][i+1];
	Q[1][j][i-2] = -Q[1][j][i+1];
	Q[2][j][i-2] =  Q[2][j][i+1];
    }
    if (ti == nx_+1 && bc_east_ == 1) {
	// Wall boundary on east
	Q[0][j][i+1] =  Q[0][j][i];
	Q[1][j][i+1] = -Q[1][j][i];
	Q[2][j][i+1] =  Q[2][j][i];
        
	Q[0][j][i+2] =  Q[0][j][i-1];
	Q[1][j][i+2] = -Q[1][j][i-1];
	Q[2][j][i+2] =  Q[2][j][i-1];
    }
    if (tj == 2 && bc_south_ == 1) {
	// Wall boundary on south
	Q[0][j-1][i] =  Q[0][j][i];
	Q[1][j-1][i] =  Q[1][j][i];
	Q[2][j-1][i] = -Q[2][j][i];
        
	Q[0][j-2][i] =  Q[0][j+1][i];
	Q[1][j-2][i] =  Q[1][j+1][i];
	Q[2][j-2][i] = -Q[2][j+1][i];
    }
    if (tj == ny_+1 && bc_north_ == 1) {
	// Wall boundary on north
	Q[0][j+1][i] =  Q[0][j][i];
	Q[1][j+1][i] =  Q[1][j][i];
	Q[2][j+1][i] = -Q[2][j][i];
        
	Q[0][j+2][i] =  Q[0][j-1][i];
	Q[1][j+2][i] =  Q[1][j-1][i];
	Q[2][j+2][i] = -Q[2][j-1][i];
    }
}


/**
  * No flow boundary conditions for the shallow water equations
  * with two ghost cells in each direction
  */
__device__ void noFlowBoundary2(float Q[3][block_height+4][block_width+4], const int nx_, const int ny_, const int boundary_conditions_type_) {
    if (boundary_conditions_type_ == 2) {
	return;
    }
    int bc_north = 1;
    int bc_east = 1;
    int bc_south = 1;
    int bc_west = 1;
    if (boundary_conditions_type_ == 3) {
	bc_north = 2;
	bc_south = 2;
    }
    else if (boundary_conditions_type_ == 4) {
	bc_east = 2;
	bc_west = 2;
    }

    noFlowBoundary2Mix(Q, nx_, ny_, bc_north, bc_east, bc_south, bc_west);
}




/**
  * Evolves the solution in time along the x axis (dimensional splitting)
  */
__device__ void evolveF1(float Q[3][block_height+2][block_width+2],
              float F[3][block_height+1][block_width+1],
              const int nx_, const int ny_,
              const float dx_, const float dt_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    //Index of cell within domain
    const int ti = blockIdx.x * blockDim.x + threadIdx.x + 1; //Skip global ghost cells, i.e., +1
    const int tj = blockIdx.y * blockDim.y + threadIdx.y + 1;
    
    if (ti > 0 && ti < nx_+1 && tj > 0 && tj < ny_+1) {
        const int i = tx + 1; //Skip local ghost cells, i.e., +1
        const int j = ty + 1;
        
        Q[0][j][i] = Q[0][j][i] + (F[0][ty][tx] - F[0][ty][tx+1]) * dt_ / dx_;
        Q[1][j][i] = Q[1][j][i] + (F[1][ty][tx] - F[1][ty][tx+1]) * dt_ / dx_;
        Q[2][j][i] = Q[2][j][i] + (F[2][ty][tx] - F[2][ty][tx+1]) * dt_ / dx_;
    }
}






/**
  * Evolves the solution in time along the x axis (dimensional splitting)
  */
__device__ void evolveF2(float Q[3][block_height+4][block_width+4],
              float F[3][block_height+1][block_width+1],
              const int nx_, const int ny_,
              const float dx_, const float dt_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;    
    
    //Index of cell within domain
    const int ti = blockIdx.x * blockDim.x + threadIdx.x + 2; //Skip global ghost cells, i.e., +2
    const int tj = blockIdx.y * blockDim.y + threadIdx.y + 2;
    
    if (ti > 1 && ti < nx_+2 && tj > 1 && tj < ny_+2) {
        const int i = tx + 2; //Skip local ghost cells, i.e., +1
        const int j = ty + 2;
        
        Q[0][j][i] = Q[0][j][i] + (F[0][ty][tx] - F[0][ty][tx+1]) * dt_ / dx_;
        Q[1][j][i] = Q[1][j][i] + (F[1][ty][tx] - F[1][ty][tx+1]) * dt_ / dx_;
        Q[2][j][i] = Q[2][j][i] + (F[2][ty][tx] - F[2][ty][tx+1]) * dt_ / dx_;
    }
}






/**
  * Evolves the solution in time along the y axis (dimensional splitting)
  */
__device__ void evolveG1(float Q[3][block_height+2][block_width+2],
              float G[3][block_height+1][block_width+1],
              const int nx_, const int ny_,
              const float dy_, const float dt_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;    
    
    //Index of cell within domain
    const int ti = blockIdx.x * blockDim.x + threadIdx.x + 1; //Skip global ghost cells, i.e., +1
    const int tj = blockIdx.y * blockDim.y + threadIdx.y + 1;
    
    if (ti > 0 && ti < nx_+1 && tj > 0 && tj < ny_+1) {
        const int i = tx + 1; //Skip local ghost cells, i.e., +1
        const int j = ty + 1;
        
        Q[0][j][i] = Q[0][j][i] + (G[0][ty][tx] - G[0][ty+1][tx]) * dt_ / dy_;
        Q[1][j][i] = Q[1][j][i] + (G[1][ty][tx] - G[1][ty+1][tx]) * dt_ / dy_;
        Q[2][j][i] = Q[2][j][i] + (G[2][ty][tx] - G[2][ty+1][tx]) * dt_ / dy_;
    }
}







/**
  * Evolves the solution in time along the y axis (dimensional splitting)
  */
__device__ void evolveG2(float Q[3][block_height+4][block_width+4],
              float G[3][block_height+1][block_width+1],
              const int nx_, const int ny_,
              const float dy_, const float dt_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    //Index of cell within domain
    const int ti = blockIdx.x * blockDim.x + threadIdx.x + 2; //Skip global ghost cells, i.e., +2
    const int tj = blockIdx.y * blockDim.y + threadIdx.y + 2;
    
    if (ti > 1 && ti < nx_+2 && tj > 1 && tj < ny_+2) {
        const int i = tx + 2; //Skip local ghost cells, i.e., +2
        const int j = ty + 2;
        
        Q[0][j][i] = Q[0][j][i] + (G[0][ty][tx] - G[0][ty+1][tx]) * dt_ / dy_;
        Q[1][j][i] = Q[1][j][i] + (G[1][ty][tx] - G[1][ty+1][tx]) * dt_ / dy_;
        Q[2][j][i] = Q[2][j][i] + (G[2][ty][tx] - G[2][ty+1][tx]) * dt_ / dy_;
    }
}










/**
  * Reconstructs a slope using the minmod limiter based on three 
  * consecutive values
  */
__device__ float minmodSlope(float left, float center, float right, float theta) {
    const float backward = (center - left) * theta;
    const float central = (right - left) * 0.5f;
    const float forward = (right - center) * theta;
    
	return 0.25f
		*copysign(1.0f, backward)
		*(copysign(1.0f, backward) + copysign(1.0f, central))
		*(copysign(1.0f, central) + copysign(1.0f, forward))
		*min( min(fabs(backward), fabs(central)), fabs(forward) );
}

__device__ float minmodRaw(float backward, float central, float forward) {

    return 0.25f
	*copysign(1.0f, backward)
	*(copysign(1.0f, backward) + copysign(1.0f, central))
	*(copysign(1.0f, central) + copysign(1.0f, forward))
	*min( min(fabs(backward), fabs(central)), fabs(forward) );
}


/**
  * Reconstructs a minmod slope for a whole block along x
  */
__device__ void minmodSlopeX(float  Q[3][block_height+4][block_width+4],
                  float Qx[3][block_height+2][block_width+2],
                  const float theta_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    //Reconstruct slopes along x axis
    for (int j=ty; j<block_height; j+=blockDim.y) {
        const int l = j + 2; //Skip ghost cells
        for (int i=tx; i<block_width+2; i+=blockDim.x) {
            const int k = i + 1;
            for (int p=0; p<3; ++p) {
                Qx[p][j][i] = 0.5f * minmodSlope(Q[p][l][k-1], Q[p][l][k], Q[p][l][k+1], theta_);
            }
        }
    }
}



/**
  * Reconstructs a minmod slope for a whole block along y
  */
__device__ void minmodSlopeY(float  Q[3][block_height+4][block_width+4],
                  float Qy[3][block_height+2][block_width+2],
                  const float theta_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    for (int j=ty; j<block_height+2; j+=blockDim.y) {
        const int l = j + 1;
        for (int i=tx; i<block_width; i+=blockDim.x) {            
            const int k = i + 2; //Skip ghost cells
            for (int p=0; p<3; ++p) {
                Qy[p][j][i] = 0.5f * minmodSlope(Q[p][l-1][k], Q[p][l][k], Q[p][l+1][k], theta_);
            }
        }
    }
}



texture<float, hipTextureType2D> windstress_X_current;
texture<float, hipTextureType2D> windstress_X_next;

texture<float, hipTextureType2D> windstress_Y_current;
texture<float, hipTextureType2D> windstress_Y_next;


/**
  * Returns the wind stress, trilinearly interpolated in space and time
  * @param wind_stress_t_ \in [0, 1] determines the temporal interpolation (0=current, 1=next)
  * @param ti_ Location of this thread along the x-axis in number of cells (NOTE: half indices)
  * @param tj_ Location of this thread along the y-axis in number of cells (NOTE: half indices)
  * @param nx_ Number of cells along x axis
  * @param ny_ Number of cells along y axis
  */
__device__ float windStressX(float wind_stress_t_, float ti_, float tj_, int nx_, int ny_) {
    //Normalize coordinates (to [0, 1])
    const int s = ti_ / float(nx_);
    const int t = tj_ / float(ny_);
    
    //Look up current and next timestep (using bilinear texture interpolation)
    float current = tex2D(windstress_X_current, s, t);
    float next = tex2D(windstress_X_next, s, t);
    
    //Interpolate in time
    return wind_stress_t_*next + (1.0f - wind_stress_t_)*current;
}

/**
  * Returns the wind stress, trilinearly interpolated in space and time
  * @param wind_stress_t_ \in [0, 1] determines the temporal interpolation (0=current, 1=next)
  * @param ti_ Location of this thread along the x-axis in number of cells (NOTE: half indices)
  * @param tj_ Location of this thread along the y-axis in number of cells (NOTE: half indices)
  * @param nx_ Number of cells along x axis
  * @param ny_ Number of cells along y axis
  */
__device__ float windStressY(float wind_stress_t_, float ti_, float tj_, int nx_, int ny_) {
    //Normalize coordinates (to [0, 1])
    const int s = ti_ / float(nx_);
    const int t = tj_ / float(ny_);
    
    //Look up current and next timestep (using bilinear texture interpolation)
    float current = tex2D(windstress_Y_current, s, t);
    float next = tex2D(windstress_Y_next, s, t);
    
    //Interpolate in time
    return wind_stress_t_*next + (1.0f - wind_stress_t_)*current;
}





__device__ float3 F_func(const float3 Q, const float g) {
    float3 F;

    F.x = Q.y;                              //hu
    F.y = Q.y*Q.y / Q.x + 0.5f*g*Q.x*Q.x;   //hu*hu/h + 0.5f*g*h*h;
    F.z = Q.y*Q.z / Q.x;                    //hu*hv/h;

    return F;
}


/**
  * Central upwind flux function
  * Takes Q = [h, hu, hv] as input, not [w, hu, hv].
  */
__device__ float3 CentralUpwindFlux(const float3 Qm, float3 Qp, const float g) {
    const float3 Fp = F_func(Qp, g);
    const float up = Qp.y / Qp.x;   // hu / h
    const float cp = sqrt(g*Qp.x); // sqrt(g*h)

    const float3 Fm = F_func(Qm, g);
    const float um = Qm.y / Qm.x;   // hu / h
    const float cm = sqrt(g*Qm.x); // sqrt(g*h)
    
    const float am = min(min(um-cm, up-cp), 0.0f); // largest negative wave speed
    const float ap = max(max(um+cm, up+cp), 0.0f); // largest positive wave speed
    
    return ((ap*Fm - am*Fp) + ap*am*(Qp-Qm))/(ap-am);
}


__device__ float3 F_func_bottom(const float3 Q, const float h, const float u, const float g) {
    float3 F;

    F.x = Q.y;                       //hu
    F.y = Q.y*u + 0.5f*g*(h*h);      //hu*u + 0.5f*g*h*h;
    F.z = Q.z*u;                     //hv*u;

    return F;
}

/**
  * Central upwind flux function
  * Takes Q = [eta, hu, hv] as input
  */
__device__ float3 CentralUpwindFluxBottom(const float3 Qm, float3 Qp, const float H, const float g) {
    const float hp = Qp.x + H;  // h = eta + H
    const float up = Qp.y / (float) hp; // hu/h
    const float3 Fp = F_func_bottom(Qp, hp, up, g);
    const float cp = sqrt(g*hp); // sqrt(g*h)

    const float hm = Qm.x + H;
    const float um = Qm.y / (float) hm;   // hu / h
    const float3 Fm = F_func_bottom(Qm, hm, um, g);
    const float cm = sqrt(g*hm); // sqrt(g*h)
    
    const float am = min(min(um-cm, up-cp), 0.0f); // largest negative wave speed
    const float ap = max(max(um+cm, up+cp), 0.0f); // largest positive wave speed
    // Related to dry zones
    // The constant is a compiler constant in the CUDA code.
    const float KPSIMULATOR_FLUX_SLOPE_EPS = 1.0e-4f;
    if ( fabs(ap - am) < KPSIMULATOR_FLUX_SLOPE_EPS ) {
	return make_float3(0.0f, 0.0f, 0.0f);
    }
    
    return ((ap*Fm - am*Fp) + ap*am*(Qp-Qm))/(ap-am);
}


/**
  *  Source terms related to bathymetry  
  */
__device__ float bottomSourceTerm2(float Q[3][block_height+4][block_width+4],
			float  Qx[3][block_height+2][block_width+2],
			float RHx[block_height+4][block_width+4],
			const float g, 
			const int p, const int q) {
    // Compansating for the smaller shmem for Qx relative to Q:
    const int pQx = p - 1;
    const int qQx = q - 2;
    
    const float hp = Q[0][q][p] + Qx[0][qQx][pQx];
    const float hm = Q[0][q][p] - Qx[0][qQx][pQx];
    // g (w - B)*B_x -> KP07 equations (3.15) and (3.16)
    // With eta: g (eta + H)*(-H_x)
    return -0.5f*g*(RHx[q][p+1] - RHx[q][p])*(hp + RHx[q][p+1] + hm + RHx[q][p]);
}

__device__ float bottomSourceTerm3(float Q[3][block_height+4][block_width+4],
			float  Qy[3][block_height+2][block_width+2],
			float RHy[block_height+4][block_width+4],
			const float g, 
			const int p, const int q) {
    // Compansating for the smaller shmem for Qy relative to Q:
    const int pQy = p - 2;
    const int qQy = q - 1;
    
    const float hp = Q[0][q][p] + Qy[0][qQy][pQy];
    const float hm = Q[0][q][p] - Qy[0][qQy][pQy];
    return -0.5f*g*(RHy[q+1][p] - RHy[q][p])*(hp + RHy[q+1][p] + hm + RHy[q][p]);
}




#endif // COMMON_CU
