#include "hip/hip_runtime.h"
/*
This OpenCL kernel implements part of the Forward Backward Linear 
numerical scheme for the shallow water equations, described in 
L. P. Røed, "Documentation of simple ocean models for use in ensemble
predictions", Met no report 2012/3 and 2012/5 .

Copyright (C) 2016  SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "common.cu"

// Finds the coriolis term based on the linear Coriolis force
// f = \tilde{f} + beta*(y-y0)
__device__ float linear_coriolis_term(const float f, const float beta,
			   const float tj, const float dy,
			   const float y_zero_reference_cell) {
    // y_0 is at the southern face of the row y_zero_reference_cell.
    float y = (tj-y_zero_reference_cell + 0.0f)*dy;
    return f + beta * y;
}


/**
  * Kernel that evolves V one step in time.
  */
extern "C" {
__global__ void computeVKernel(
        //Discretization parameters
        int nx_, int ny_,
        float dx_, float dy_, float dt_,
    
        //Physical parameters
        float g_, //< Gravitational constant
        float f_, //< Coriolis coefficient
	float beta_, //< Coriolis force f_ + beta_*(y-y0)
	float y_zero_reference_cell_, // the cell row representing y0 (y0 at southern face)
        float r_, //< Bottom friction coefficient
    
        //Data
        float* H_ptr_, int H_pitch_,
        float* U_ptr_, int U_pitch_,
        float* V_ptr_, int V_pitch_,
        float* eta_ptr_, int eta_pitch_,
    
        // Wind stress parameters
        float wind_stress_t_) {
        
    __shared__ float H_shared[block_height+1][block_width];
    __shared__ float U_shared[block_height+1][block_width+1];
    __shared__ float eta_shared[block_height+1][block_width];

    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    //Index of block within domain
    const int bx = blockDim.x * blockIdx.x;
    const int by = blockDim.y * blockIdx.y;

    //Index of cell within domain
    const int ti = bx + tx;
    const int tj = by + ty;

    //Compute pointer to current row in the V array
    float* const V_row = (float*) ((char*) V_ptr_ + V_pitch_*tj);

    //Read current V
    float V_current = 0.0f;
    if (ti < nx_ && tj < ny_+1) {
        V_current = V_row[ti];
    }

    //Read H and eta into shared memory
    for (int j=ty; j<block_height+1; j+=blockDim.y) {
        const int l = by + j - 1;
        
        //Compute the pointer to current row in the H and eta arrays
        float* const H_row = (float*) ((char*) H_ptr_ + H_pitch_*l);
        float* const eta_row = (float*) ((char*) eta_ptr_ + eta_pitch_*l);
        
        for (int i=tx; i<block_width; i+=blockDim.x) {
            const int k = bx + i;
            if (k < nx_ && l >= 0 && l < ny_) {
                H_shared[j][i] = H_row[k];
                eta_shared[j][i] = eta_row[k];
            }
            else {
                H_shared[j][i] = 0.0f;
                eta_shared[j][i] = 0.0f;
            }
        }
    }

    //Read U into shared memory
    for (int j=ty; j<block_height+1; j+=blockDim.y) {
        const int l = by + j - 1;
        
        //Compute the pointer to current row in the V array
        float* const U_row = (float*) ((char*) U_ptr_ + U_pitch_*l);
        
        for (int i=tx; i<block_width+1; i+=blockDim.x) {
            const int k = bx + i;
            if (k < nx_+1 && l >= 0 && l < ny_) {
                U_shared[j][i] = U_row[k];
            }
            else {
                U_shared[j][i] = 0.0f;
            }
        }
    }

    //Make sure all threads have read into shared mem
    __syncthreads();

    //Reconstruct H at the V position
    float H_m = 0.5f*(H_shared[ty][tx] + H_shared[ty+1][tx]);

    // Coriolis forces at V position and U positions
    float f_v   = linear_coriolis_term(f_, beta_, tj,      dy_, y_zero_reference_cell_);
    float f_u_p = linear_coriolis_term(f_, beta_, tj+0.5f, dy_, y_zero_reference_cell_);
    float f_u_m = linear_coriolis_term(f_, beta_, tj-0.5f, dy_, y_zero_reference_cell_); 
    
    //Reconstruct f*U at the V position
    float fU_m;
    if (ti==0) {
	// Using Coriolis at V postiion
        fU_m = 0.5f*f_v*(U_shared[ty][tx+1] + U_shared[ty+1][tx+1]);
    }
    else if (ti==nx_-1) {
	// Using Coriolis at V postiion
        fU_m = 0.5f*f_v*(U_shared[ty][tx] + U_shared[ty+1][tx]);
    }
    else {
        fU_m = 0.25f*( f_u_m*(U_shared[ty  ][tx] + U_shared[ty  ][tx+1])
		     + f_u_p*(U_shared[ty+1][tx] + U_shared[ty+1][tx+1]) );
    }

    //Calculate the friction coefficient
    float B = H_m/(H_m + r_*dt_);

    //Calculate the gravitational effect
    float P = g_*H_m*(eta_shared[ty][tx] - eta_shared[ty+1][tx])/dy_;

    //FIXME Check coordinates (ti_, tj_) here!!!
    //TODO Check coordinates (ti_, tj_) here!!!
    //WARNING Check coordinates (ti_, tj_) here!!!
    float Y = windStressY(wind_stress_t_, ti+0.5, tj, nx_, ny_);

    //Compute the V at the next timestep
    float V_next = B*(V_current + dt_*(-fU_m + P + Y) );

    //Write to main memory
    if (ti < nx_ && tj > 0 && tj < ny_ ) {
        V_row[ti] = V_next;
    }

    // TODO:
    // Currently, boundary conditions are individual kernels.
    // They should be moved to be within-kernel functions.
}
} // extern "C" 