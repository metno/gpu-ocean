#include "hip/hip_runtime.h"
/*
This software is part of GPU Ocean. 

Copyright (C) 2018 SINTEF Digital
Copyright (C) 2018 Norwegian Meteorological Institute

This CUDA kernel implements a selection of drift trajectory algorithms.

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/



/**
  * Kernel that evolves drifter positions along u and v.
  */
  
//Code relating to wind-data

texture<float, hipTextureType2D> wind_X_current;
texture<float, hipTextureType2D> wind_X_next;

texture<float, hipTextureType2D> wind_Y_current;
texture<float, hipTextureType2D> wind_Y_next;

__device__ float windX(const float wind_t_, 
                       const float drifter_pos_x_, const float drifter_pos_y_, 
                       const float domain_size_x_, const float domain_size_y_) {
    
    //Normalize coordinates (to [0, 1])
    const float s = drifter_pos_x_ / domain_size_x_;
    const float t = drifter_pos_y_ / domain_size_y_;
    
    //Look up current and next timestep (using bilinear texture interpolation)
    const float current = tex2D(wind_X_current, s, t);
    const float next = tex2D(wind_X_next, s, t);
    
    //Interpolate in time
    return wind_t_*next + (1.0f - wind_t_)*current;
}

__device__ float windY(const float wind_t_, 
                       const float drifter_pos_x_, const float drifter_pos_y_, 
                       const float domain_size_x_, const float domain_size_y_) {
    
    //Normalize coordinates (to [0, 1])
    const float s = drifter_pos_x_ / domain_size_x_;
    const float t = drifter_pos_y_ / domain_size_y_;
    
    //Look up current and next timestep (using bilinear texture interpolation)
    const float current = tex2D(wind_Y_current, s, t);
    const float next = tex2D(wind_Y_next, s, t);
    
    //Interpolate in time
    return wind_t_*next + (1.0f - wind_t_)*current;
}



__device__ float waterVelocityU(
        float* eta_ptr_, const int eta_pitch_,
        float* hu_ptr_, const int hu_pitch_,
        float* Hm_ptr_, const int Hm_pitch_,
        const int cell_id_x, const int cell_id_y) {
    
    // Read the water velocity from global memory
    float* const eta_row_y = (float*) ((char*) eta_ptr_ + eta_pitch_*cell_id_y);
    float* const Hm_row_y = (float*) ((char*) Hm_ptr_ + Hm_pitch_*cell_id_y);
    float const h = Hm_row_y[cell_id_x] + eta_row_y[cell_id_x];

    float* const hu_row = (float*) ((char*) hu_ptr_ + hu_pitch_*cell_id_y);
    
    float const u = hu_row[cell_id_x]/h;
    
    return u;
}

__device__ float waterVelocityV(
        float* eta_ptr_, const int eta_pitch_,
        float* hv_ptr_, const int hv_pitch_, 
        float* Hm_ptr_, const int Hm_pitch_,
        const int cell_id_x, const int cell_id_y) {
    
    // Read the water velocity from global memory
    float* const eta_row_y = (float*) ((char*) eta_ptr_ + eta_pitch_*cell_id_y);
    float* const Hm_row_y = (float*) ((char*) Hm_ptr_ + Hm_pitch_*cell_id_y);
    float const h = Hm_row_y[cell_id_x] + eta_row_y[cell_id_x];

    float* const hv_row = (float*) ((char*) hv_ptr_ + hv_pitch_*cell_id_y);
    
    float const v = hv_row[cell_id_x]/h;
    
    return v;
}

extern "C" {
__global__ void passiveDrifterKernel(
        //Discretization parameters
        const int nx_, const int ny_,
        const float dx_, const float dy_, const float dt_,

        const int x_zero_reference_cell_, // the cell column representing x0 (x0 at western face)
        const int y_zero_reference_cell_, // the cell row representing y0 (y0 at southern face)
        
        // Data
        float* eta_ptr_, const int eta_pitch_,
        float* hu_ptr_, const int hu_pitch_,
        float* hv_ptr_, const int hv_pitch_,
        // H should be read from buffer, but for now we use a constant value
        float* Hm_ptr_, const int Hm_pitch_,

        const int periodic_north_south_,
        const int periodic_east_west_,
        
        const int num_drifters_,
        float* drifters_positions_, const int drifters_pitch_,
        const float sensitivity_,
        const float wind_t_, 
        const float wind_drift_factor_) 
        {

    //Index of thread within block (only needed in one dim)
    const int tx = threadIdx.x;
        
    //Index of block within domain (only needed in one dim)
    const int bx = blockDim.x * blockIdx.x;
        
    //Index of cell within domain (only needed in one dim)
    const int ti = bx + tx;
    
    if (ti < num_drifters_ + 1) {
        // Obtain pointer to our particle:
        float* drifter = (float*) ((char*) drifters_positions_ + drifters_pitch_*ti);
        float drifter_pos_x = drifter[0];
        float drifter_pos_y = drifter[1];
        
        // Find cell ID for the cell in which our particle is
        int const cell_id_x = (int)(ceil(drifter_pos_x/dx_) + x_zero_reference_cell_);
        int const cell_id_y = (int)(ceil(drifter_pos_y/dy_) + y_zero_reference_cell_);
        
        float const frac_x = drifter_pos_x / dx_ - floor(drifter_pos_x / dx_);
        float const frac_y = drifter_pos_y / dy_ - floor(drifter_pos_y / dy_);
        
        const int cell_id_x0 = frac_x < 0.5f ? cell_id_x - 1 : cell_id_x;
        const float x_factor = frac_x < 0.5f ? frac_x + 0.5f : frac_x - 0.5f; 
        const int cell_id_x1 = cell_id_x0 + 1;

        const int cell_id_y0 = frac_y < 0.5f ? cell_id_y - 1 : cell_id_y;
        const float y_factor = frac_y < 0.5f ? frac_y + 0.5f : frac_y - 0.5f; 
        const int cell_id_y1 = cell_id_y0 + 1;
                
        float const u_x0y0 = waterVelocityU(eta_ptr_, eta_pitch_,hu_ptr_, hu_pitch_,Hm_ptr_, Hm_pitch_, cell_id_x0, cell_id_y0);
        float const u_x1y0 = waterVelocityU(eta_ptr_, eta_pitch_,hu_ptr_, hu_pitch_,Hm_ptr_, Hm_pitch_, cell_id_x1, cell_id_y0);
        float const u_x0y1 = waterVelocityU(eta_ptr_, eta_pitch_,hu_ptr_, hu_pitch_,Hm_ptr_, Hm_pitch_, cell_id_x0, cell_id_y1);
        float const u_x1y1 = waterVelocityU(eta_ptr_, eta_pitch_,hu_ptr_, hu_pitch_,Hm_ptr_, Hm_pitch_, cell_id_x1, cell_id_y1);
        
        float const v_x0y0 = waterVelocityV(eta_ptr_, eta_pitch_,hv_ptr_, hv_pitch_,Hm_ptr_, Hm_pitch_, cell_id_x0, cell_id_y0);
        float const v_x1y0 = waterVelocityV(eta_ptr_, eta_pitch_,hv_ptr_, hv_pitch_,Hm_ptr_, Hm_pitch_, cell_id_x1, cell_id_y0);
        float const v_x0y1 = waterVelocityV(eta_ptr_, eta_pitch_,hv_ptr_, hv_pitch_,Hm_ptr_, Hm_pitch_, cell_id_x0, cell_id_y1);
        float const v_x1y1 = waterVelocityV(eta_ptr_, eta_pitch_,hv_ptr_, hv_pitch_,Hm_ptr_, Hm_pitch_, cell_id_x1, cell_id_y1);
        
        float const u_y0 = (1-x_factor)*u_x0y0 + x_factor * u_x1y0; 
        float const u_y1 = (1-x_factor)*u_x0y1 + x_factor * u_x1y1; 
        
        float const v_y0 = (1-x_factor)*v_x0y0 + x_factor * v_x1y0; 
        float const v_y1 = (1-x_factor)*v_x0y1 + x_factor * v_x1y1;
        
        float u = (1-y_factor)*u_y0 + y_factor *u_y1;
        float v = (1-y_factor)*v_y0 + y_factor *v_y1;
        
        if (wind_drift_factor_) {
            u = u + windX(wind_t_, drifter_pos_x, drifter_pos_y, nx_*dx_, ny_*dy_) * wind_drift_factor_;
            v = v + windY(wind_t_, drifter_pos_x, drifter_pos_y, nx_*dx_, ny_*dy_) * wind_drift_factor_;
        }
        
        // Move drifter
        drifter_pos_x += sensitivity_*u*dt_;
        drifter_pos_y += sensitivity_*v*dt_;
            
        // Ensure boundary conditions
        if (periodic_east_west_ && (drifter_pos_x < 0)) {
            drifter_pos_x += + nx_*dx_;
        }
        if (periodic_east_west_ && (drifter_pos_x > nx_*dx_)) {
            drifter_pos_x -= nx_*dx_;
        }
        if (periodic_north_south_ && (drifter_pos_y < 0)) {
            drifter_pos_y += ny_*dy_;
        }
        if (periodic_north_south_ && (drifter_pos_y > ny_*dy_)) {
            drifter_pos_y -= ny_*dy_;
        }

        // Write to global memory
        drifter[0] = drifter_pos_x;
        drifter[1] = drifter_pos_y;
    }
}
} // extern "C"
    

extern "C" {
__global__ void enforceBoundaryConditions(
        //domain parameters
        float domain_size_x_, float domain_size_y_,

        int periodic_north_south_,
        int periodic_east_west_,
        
        int num_drifters_,
        float* drifters_positions_, int drifters_pitch_) {
    
    //Index of drifter (only needed in one dimension)
    const int ti = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (ti < num_drifters_ + 1) {
        // Obtain pointer to our particle:
        float* drifter = (float*) ((char*) drifters_positions_ + drifters_pitch_*ti);
        float drifter_pos_x = drifter[0];
        float drifter_pos_y = drifter[1];

        // Ensure boundary conditions
        if (periodic_east_west_ && (drifter_pos_x < 0)) {
            drifter_pos_x += + domain_size_x_;
        }
        if (periodic_east_west_ && (drifter_pos_x > domain_size_x_)) {
            drifter_pos_x -= domain_size_x_;
        }
        if (periodic_north_south_ && (drifter_pos_y < 0)) {
            drifter_pos_y += domain_size_y_;
        }
        if (periodic_north_south_ && (drifter_pos_y > domain_size_y_)) {
            drifter_pos_y -= domain_size_y_;
        }

        // Write to global memory
        drifter[0] = drifter_pos_x;
        drifter[1] = drifter_pos_y;
    }
}
} // extern "C"
